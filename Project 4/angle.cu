//*****************************
// Bartu Atabek 21602229
// CS 426: Parallel Computing
// Project IV
//*****************************

// #define imin(a,b) (a<b?a:b)
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

/* Array generator code which generates a random integer array with N elements. */
int* arrayGenerator(int N) {
	if (N <= 0) {
		return NULL;
	}

	int* vector = (int*) malloc(N * sizeof(int));
	for (int i = 0; i < N; i++) {
		vector[i] = rand() % 100;
	}
	return vector;
}

/* Reads the first line of the input file and returns it as N. */
int getArraySize(char* filename) {
	FILE* file;
	int numOfArraySize = 0;

	if ((file = fopen(filename, "r")) == NULL) {
		fprintf(stderr, "Error: failed to open file %s\n", filename);
		return numOfArraySize;
	}

	fscanf(file, "%d", &numOfArraySize);
	fclose(file);
	return numOfArraySize;
}

/* Reads the inputted file and returns input vectors. */
void readFile(char* filename, int arrSize, int** vector1, int** vector2) {
	int a = 0;
	FILE* file = fopen(filename, "r");
	fscanf(file, "%d", &a);

	int x = 0;
	int i = 0;
	int j = 0;
	while (!feof(file)) {
		fscanf(file, "%d", &x);
		if (i < arrSize) {
			(*vector1)[i] = x;
		}
		if (i >= arrSize && i < 2 * arrSize) {
			(*vector2)[j] = x;
			j++;
		}
		i++;
	}
	fclose(file);
}

/* Finds the angle between two vectors. */
double findAngle(int N, int* vector1, int* vector2) {
	float nominator = 0;
	double length1 = 0;
	double length2 = 0;
	double denominator = 0;
	double result = 0;
	double value = 180.0 / M_PI;

	for (int i = 0; i < N; i++) {
		nominator += vector1[i] * vector2[i];
	}

	for (int i = 0; i < N; i++) {
		length1 += pow(vector1[i], 2);
		length2 += pow(vector2[i], 2);
	}
	length1 = sqrt(length1);
	length2 = sqrt(length2);
	denominator = length1 * length2;

	result = nominator / denominator;
	result = atan(result) * value;
	return result;
}

__global__ void compute(int N, int threadsPerBlock, int *d_vector1, int *d_vector2, float *d_vector3) {
	extern __shared__ float temp[];
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	// Calculate the nominator using dot product of two vectors
	temp[threadIdx.x] = d_vector1[index] * d_vector2[index];

	// Synchronize threads
	__syncthreads();

	// Accumulate the results
	if (0 == threadIdx.x) {
		int sum = 0;
		for (int i = 0; i < threadsPerBlock; i++)
			sum += temp[i];
		atomicAdd(d_vector3, sum);
		__syncthreads();
	}
	__syncthreads();

	// Calculate the denominator using ||v1|| * ||v2||
	// Calculate the sqrt of first vector
	temp[threadIdx.x] = powf(d_vector1[index], 2);

	// Synchronize threads
	__syncthreads();

	// Accumulate the results
	if (0 == threadIdx.x) {
		int sum = 0;
		for (int i = 0; i < threadsPerBlock; i++)
			sum += temp[i];
		atomicAdd(d_vector3 + 1, sum);
		__syncthreads();
	}
	__syncthreads();

	// Calculate the sqrt of second vector
	temp[threadIdx.x] = powf(d_vector2[index], 2);

	// Synchronize threads
	__syncthreads();

	// Accumulate the results
	if (0 == threadIdx.x) {
		int sum = 0;
		for (int i = 0; i < threadsPerBlock; i++)
			sum += temp[i];
		atomicAdd(d_vector3 + 2, sum);
		__syncthreads();
	}
	__syncthreads();
}

int main(int argc, char **argv) {
	// Info
	int N;
	int threadsPerBlock;
	int blocksPerGrid;

	// Results
	double CPU_result, GPU_result;

	// Time measure properties for CPU
	clock_t start, end;
	double time_for_arr_gen, time_for_cpu_func, time_for_host_to_device, time_for_device_to_host, time_for_kernel_exe;

	// Input/ouput vectors of the host and kernel
	int *vector1, *vector2, *d_vector1, *d_vector2;
	float *output, *d_output;

	if (argc == 3) {
		// To measure time for GPU
		hipEvent_t start_gpu, stop_gpu;
		hipEventCreate(&start_gpu);
		hipEventCreate(&stop_gpu);

		N = atoi(argv[1]);
		threadsPerBlock = atoi(argv[2]);
		blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

		// Initialize time t for pseudo random numbers
		time_t t;

		// Initialize input vectors and output
		start = clock();
		srand((unsigned) time(&t));
		vector1 = arrayGenerator(N);
		vector2 = arrayGenerator(N);
		output = (float*) malloc(3 * sizeof(float));
		output[0] = 0; output[1] = 0; output[2] = 0;

		// Memory allocation for device members
		hipMalloc((void**)&d_vector1, N * sizeof(int));
		hipMalloc((void**)&d_vector2, N * sizeof(int));
		hipMalloc((void**)&d_output, 3 * sizeof(float));
		end = clock();
		time_for_arr_gen = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Host to device transfer
		start = clock();
		hipMemcpy(d_vector1, vector1, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_vector2, vector2, N * sizeof(int), hipMemcpyHostToDevice);
		end = clock();
		time_for_host_to_device = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Run host function
		start = clock();
		CPU_result = findAngle(N, vector1, vector2);
		end = clock();
		time_for_cpu_func = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Run kernel function
		start = clock();
		compute<<< blocksPerGrid, threadsPerBlock, (threadsPerBlock * sizeof(float)) >>>(N, threadsPerBlock, d_vector1, d_vector2, d_output);

		hipDeviceSynchronize();
		end = clock();
		time_for_kernel_exe = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Device to host transfer
		start = clock();
		hipMemcpy(output, d_output, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		end = clock();
		time_for_device_to_host = ((double)(end - start)) / CLOCKS_PER_SEC;

		output[1] = sqrt(output[1]);
		output[2] = sqrt(output[2]);
		float nominator = output[0];
		float denominator = output[1] * output[2];
		GPU_result = nominator / denominator;
		double value = 180.0 / M_PI;
		GPU_result = atan(GPU_result) * value;
	}
	else if (argc == 4) {
		// To measure time for GPU
		hipEvent_t start_gpu, stop_gpu;
		hipEventCreate(&start_gpu);
		hipEventCreate(&stop_gpu);

		// Read filename & get size of input
		char *filename = argv[3];
		int numOfArraySize = getArraySize(filename);
		N = numOfArraySize;
		threadsPerBlock = atoi(argv[2]);
		blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

		// Initialize input vectors and output
		start = clock();
		vector1 = (int*) malloc(N * sizeof(int));
		vector2 = (int*) malloc(N * sizeof(int));
		output = (float*) malloc(3 * sizeof(float));
		readFile(filename, numOfArraySize, &vector1, &vector2);
		output[0] = 0; output[1] = 0; output[2] = 0;

		// Memory allocation for device members
		hipMalloc((void**)&d_vector1, N * sizeof(int));
		hipMalloc((void**)&d_vector2, N * sizeof(int));
		hipMalloc((void**)&d_output, 3 * sizeof(float));
		end = clock();
		time_for_arr_gen = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Host to device transfer
		start = clock();
		hipMemcpy(d_vector1, vector1, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_vector2, vector2, N * sizeof(int), hipMemcpyHostToDevice);
		end = clock();
		time_for_host_to_device = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Run host function
		start = clock();
		CPU_result = findAngle(N, vector1, vector2);
		end = clock();
		time_for_cpu_func = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Run kernel function
		start = clock();
		compute<<< blocksPerGrid, threadsPerBlock, (threadsPerBlock * sizeof(float)) >>>(N, threadsPerBlock, d_vector1, d_vector2, d_output);

		hipDeviceSynchronize();
		end = clock();
		time_for_kernel_exe = ((double)(end - start)) / CLOCKS_PER_SEC;

		// Device to host transfer
		start = clock();
		hipMemcpy(output, d_output, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		end = clock();
		time_for_device_to_host = ((double)(end - start)) / CLOCKS_PER_SEC;

		output[1] = sqrt(output[1]);
		output[2] = sqrt(output[2]);
		float nominator = output[0];
		float denominator = output[1] * output[2];
		GPU_result = nominator / denominator;
		double value = 180.0 / M_PI;
		GPU_result = atan(GPU_result) * value;
	}
	else {
		fprintf(stderr, "Usage: ./a N threadsPerBlock (optional) filename.txt.\n");
		return -1;
	}

	// Display results
	printf("Info \n");
	printf("--------------------\n");
	printf("Number of Elements: %d \n", N);
	printf("Number of threads per block: %d \n", threadsPerBlock);
	printf("Number of blocks will be created: %d \n", (N + threadsPerBlock - 1) / threadsPerBlock);

	printf("Time \n");
	printf("--------------------\n");
	printf("Time for the array generation: %f ms \n", time_for_arr_gen);
	printf("Time for the CPU function: %f ms \n", time_for_cpu_func);
	printf("Time for the Host to Device transfer: %f ms \n", time_for_host_to_device / 1000);
	printf("Time for the kernel execution: %f ms \n", time_for_kernel_exe / 1000);
	printf("Time for the Device to Host transfer: %f ms \n", time_for_device_to_host / 1000);
	printf("Total execution time for GPU: %f ms \n", (time_for_host_to_device + time_for_kernel_exe) / 1000);

	printf("Results \n");
	printf("--------------------\n");
	printf("CPU result: %.3f \n", CPU_result);
	printf("GPU result: %.3f \n", GPU_result);

	free(vector1);
	free(vector2);
	free(output);
	hipFree(d_vector1);
	hipFree(d_vector2);
	hipFree(d_output);
	return 0;
}
